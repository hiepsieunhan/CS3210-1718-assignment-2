/**
* 
* Matrix Multiplication - CUDA for GPUs
*
* CS3210
*
**/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <assert.h>

#define BLOCK_SIZE 32

int size;

typedef struct
{
    int width;
    int height;
    int stride;
    float* elements;
} matrix;


long long wall_clock_time()
{
#ifdef __linux__
    struct timespec tp;
    clock_gettime(CLOCK_REALTIME, &tp);
    return (long long)(tp.tv_nsec + (long long)tp.tv_sec * 1000000000ll);
#else
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (long long)(tv.tv_usec * 1000 + (long long)tv.tv_sec * 1000000000ll);
#endif
}

/**
* Allocates memory for a matrix of size SIZE
* The memory is allocated row-major order, i.e. 
*  elements from the same row are allocated at contiguous 
*  memory addresses.
**/
void allocate_matrix(matrix* m)
{
    hipError_t rc;
    
    m->width = size;
    m->height = size;
    m->stride = size;

    // allocate all matrix elements in one array of continuous addresses
    rc = hipMallocManaged((void**)&(m->elements), sizeof(float) * size * size);
    if (rc != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(rc));
        exit(1);
    }
}

/**
* Free the memory allocated for a matrix.
**/
void free_matrix(matrix* m) {
    hipFree(m->elements);
}

/**
* Initializes the elements of the matrix with
* random values between 0 and 9
**/
void init_matrix(matrix m)
{
    int i, j;
    
    for (i = 0; i < size; i++)
        for (j = 0; j < size; j++)
        {
            m.elements[i * m.width + j] = rand() % 10;
        }
}

/**
* Initializes the elements of the matrix with
* element 0.
**/
void init_matrix_zero(matrix m)
{
    int i, j;
    
    for (i = 0; i < size; i++)
        for (j = 0; j < size; j++)
        {
            m.elements[i * m.width + j] = 0.0;
        }
}

/**
* Get element  at row, col of sub matrix
*/
__device__ float get_element(const matrix a, int row, int col, int block_x, int block_y) {
    if (block_x * BLOCK_SIZE + row >= size || block_y * BLOCK_SIZE + col >= size) 
        return 0;
    return a.elements[row * a.stride + col];
}

__device__ matrix get_sub_matrix(matrix a, int row, int col) {
    matrix a_sub;
    a_sub.width = BLOCK_SIZE;
    a_sub.height = BLOCK_SIZE;
    a_sub.stride = a.stride;
    a_sub.elements = &a.elements[a.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
    return a_sub;
}

/**
* Multiplies matrix @a with matrix @b storing
* the result in matrix @result
* 
* The multiplication algorithm is the O(n^3) 
* algorithm
*/
void mm(matrix a, matrix b, matrix result)
{
    int i, j, k;
    
    // Do the multiplication
    for (i = 0; i < size; i++)
        for (j = 0; j < size; j++) {
            float sum = 0;
            for(k = 0; k < size; k++)            
                sum += a.elements[i * a.width + k] * b.elements[k * b.width + j];
            result.elements[i * result.width + j] = sum;
        }
}

/**
* Each kernel computes the result element (i,j).
*/
__global__ void mm_kernel(matrix a, matrix b, matrix result, int size)
{
    // index in the original matric
    int g_row = blockIdx.x * blockDim.x + threadIdx.x; 
    int g_col = blockIdx.y * blockDim.y + threadIdx.y;
    int m, e;

    // block index
    int block_row = blockIdx.x;
    int block_col = blockIdx.y;

    float result_value = 0;
    // thread index
    int row = threadIdx.x;
    int col = threadIdx.y;

    int num_block = (a.width + BLOCK_SIZE - 1) / BLOCK_SIZE;
    for (m = 0; m < num_block; m++) {
        matrix a_sub = get_sub_matrix(a, block_row, m);
        matrix b_sub = get_sub_matrix(b, m, block_col);

        // Shared memory to store a_sub and b_sub
        __shared__ float a_s[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float b_s[BLOCK_SIZE][BLOCK_SIZE];

        a_s[row][col] = get_element(a_sub, row, col, block_row, m);
        b_s[row][col] = get_element(a_sub, row, col, m, block_col);
        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();

        // Multiply a_sub and b_sub together
        for (e = 0; e < BLOCK_SIZE; ++e) {
            result_value += a_s[row][e] * b_s[e][col];
        }
        // Synchronize to make sure that the preceding computation is done
        // before loading two new sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    if (g_row <= size && g_col <= size)
        result.elements[g_row * result.width + g_col] = result_value;
}

void print_matrix(matrix m)
{
    int i, j;
    
    for (i = 0; i < size; i++)
    {
        printf("row %4d: ", i);
        for (j = 0; j < size; j++)
            printf("%6.2f  ", m.elements[i * m.width + j]);
        printf("\n");
    }
}



void work()
{
    matrix a, b, result1, result2;
    long long before, after;
    int correct, i, j, dim;
    hipError_t rc;

    // Allocate memory for matrices
    allocate_matrix(&a);
    allocate_matrix(&b);
    allocate_matrix(&result1);
    allocate_matrix(&result2);	

    // Initialize matrix elements
    init_matrix(a);
    init_matrix(b);

    // Perform sequential matrix multiplication
    before = wall_clock_time();
    mm(a, b, result1);
    after = wall_clock_time();
        fprintf(stderr, "Matrix multiplication on CPU took %1.2f seconds\n", ((float)(after - before))/1000000000);

    // Perform CUDA matrix  multiplication
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);			// a block of 32 x 32 CUDA threads
    dim = (size % BLOCK_SIZE == 0) ? size / BLOCK_SIZE : size / BLOCK_SIZE + 1; 
    dim3 grid(dim, dim);	// a grid of CUDA thread blocks
    before = wall_clock_time();
    mm_kernel<<<grid, block>>>(a, b, result2, size);
    hipDeviceSynchronize();
    after = wall_clock_time();
    fprintf(stderr, "Matrix multiplication on GPU took %1.2f seconds\n", ((float)(after - before))/1000000000);

    // was there any error?
        rc = hipGetLastError();
        if (rc != hipSuccess)
                printf("Last CUDA error %s\n", hipGetErrorString(rc));

    // Compare the results
    correct = 1;
    for (i = 0; correct && i < size; i++)
        for (j = 0; j < size; j++)
            if (result1.elements[i * result1.width + j] != result2.elements[i * result2.width + j]) {
                correct = 0;
                break;
            }

    if (correct)
        printf("The result matrices are identical!\n");
    else
        printf("Difference in result matrices at element (%d, %d)!\n", i, j);

    free_matrix(&a);
    free_matrix(&b);
    free_matrix(&result1);
    free_matrix(&result2);
}


int main(int argc, char ** argv)
{
    srand(0); 

    printf("Usage: %s <size>\n", argv[0]);
    
    if (argc >= 2)
        size = atoi(argv[1]);
    else
        size = 1024;
        
    fprintf(stderr,"Sequential matrix multiplication of size %d\n", size);
    
    // Multiply the matrices
    work();

    return 0;
}
